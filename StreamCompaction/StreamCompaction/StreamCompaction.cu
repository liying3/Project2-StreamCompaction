#include "hip/hip_runtime.h"
#include "StreamCompaction.h"
#include <iostream>

using namespace std;

__global__ void prefixSumNaive(int *in, int *out, int n, int base)
{
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	
	if (tx < n)
	{
		if (tx >= base)
			out[tx] = in[tx-base] + in[tx];
		else
			out[tx] = in[tx];
	}
}

void prefixSumNaiveWarpper(int *A, int *R, int N)
{
	int *in, *out;
	hipMalloc((void**)&in, N * sizeof(int));
	hipMemset(in, 0, sizeof(int));
	hipMemcpy(in+1 , A, (N-1)* sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&out, N * sizeof(int));
	hipMemset(out, 0, sizeof(int));
	hipMemcpy(out+1 , A, (N-1)* sizeof(int), hipMemcpyHostToDevice);

	float time;
	hipEvent_t startTime, endTime;
	hipEventCreate(&startTime);
	hipEventRecord(startTime, 0);

	int base = 1;
	int	loops = log((double)BlockSize) / log(2.0);

	for (int i = 1; i <= loops; i++)
	{
		int *tem = in;
		in = out;
		out = in;

		prefixSumNaive<<<1, BlockSize>>>(in, out, N, base);
		hipDeviceSynchronize();
		base *= 2;
	}
	hipEventCreate(&endTime);
	hipEventRecord(endTime, 0);
	hipEventSynchronize(endTime);

	hipEventElapsedTime(&time, startTime, endTime);
	hipEventDestroy(startTime);
	hipEventDestroy(endTime);

	cout << "GPU(Naive): " << time << " ms" << endl;

	hipMemcpy(R, out, N*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(in);
	hipFree(out);
}

__global__ void prefixSumSharedMem(int *in, int *out, int n, int *blockSum, int base)
{
	int x = threadIdx.x;
	int tx = x + (blockIdx.x * blockDim.x);

	if (tx < n)
	{
		 __shared__ int outS[BlockSize];
		 __shared__ int inS[BlockSize];
		
		inS[x] = in[tx];
		__syncthreads();
		
		if (threadIdx.x >= base)
			outS[x] = inS[x-base] + inS[x];
		else
			outS[x] = inS[x];

		__syncthreads();

		out[tx] = outS[x];

		if (x == BlockSize-1)
			blockSum[blockIdx.x+1] = outS[x];
	}
}

__global__ void addBlockSum2All(int *in, int *out, int n, int *blockSum)
{
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);

	 if (tx < n)
	 {
		 in[tx] += blockSum[blockIdx.x];
	 }
}

void prefixSumSharedMemWarpper(int *A, int *R, int N)
{
	int *in, *out;
	hipMalloc((void**)&in, N * sizeof(int));
	hipMemset(in, 0, sizeof(int));
	hipMemcpy(in+1 , A, (N-1)* sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&out, N * sizeof(int));
	hipMemset(out, 0, sizeof(int));
	hipMemcpy(out+1 , A, (N-1)* sizeof(int), hipMemcpyHostToDevice);

	int gridSize = ((int)(ceil(N / (float)BlockSize)));
	int *blockSum;
	hipMalloc((void**)&blockSum, (gridSize+1)*sizeof(int));
	hipMemset(blockSum, 0, sizeof(int));
	
	int *blockSumOut;
	hipMalloc((void**)&blockSumOut, (gridSize+1)*sizeof(int));
	hipMemset(blockSumOut, 0, sizeof(int));
	
	float time;
	hipEvent_t startTime, endTime;
	hipEventCreate(&startTime);
	hipEventRecord(startTime, 0);

	int base = 1;
	int	loops= log((double)BlockSize) / log(2.0);

	for (int i = 1; i <= loops; i++)
	{
		int *tem = in;
		in = out;
		out = in;

		prefixSumSharedMem<<<gridSize, BlockSize>>>(in, out, N, blockSumOut, base);
		hipDeviceSynchronize();
		base *= 2;
	}
	
	if (gridSize > 1)
	{
		base = 1;
		loops= log((double)gridSize) / log(2.0);

		for (int i = 1; i <= loops; i++)
		{
			int *tem = blockSum;
			blockSum = blockSumOut;
			blockSumOut = tem;

			prefixSumNaive<<<1, BlockSize>>>(blockSum, blockSumOut, gridSize, base);
			hipDeviceSynchronize();
			base *= 2;
		}

		int *tem = in;
		in = out;
		out = tem;
		
		addBlockSum2All<<<gridSize, BlockSize>>>(in, out, N, blockSum);
		hipDeviceSynchronize();
	}
	hipEventCreate(&endTime);
	hipEventRecord(endTime, 0);
	hipEventSynchronize(endTime);

	hipEventElapsedTime(&time, startTime, endTime);
	hipEventDestroy(startTime);
	hipEventDestroy(endTime);
	cout << "GPU(Shared Memory): " << time << " ms" << endl;

	hipMemcpy(R, out, N*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(in);
	hipFree(out);
	hipFree(blockSum);
	hipFree(blockSumOut);
}

__global__ void scatter(int *A, int *R, int n)
{
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	
	if (tx < n)
	{
		if (A[tx] > 0)
			R[tx] = 1;
		else
			R[tx] = 0;
	}
}

__global__ void streamCompact(int *A, int *sum, int *R, int n)
{
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	
	if (tx < n)
	{
		if (A[tx] > 0)
			R[sum[tx]] = A[tx];
	}
}

void streamCompactionWarpper(int *A, int *R, int N)
{
	int *Ad, *scatterVd;
	hipMalloc((void**)&Ad, N * sizeof(int));
	hipMemcpy(Ad, A, N* sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&scatterVd, N * sizeof(int));

	float time;
	hipEvent_t startTime, endTime;
	hipEventCreate(&startTime);
	hipEventRecord(startTime, 0);

	int gridSize = ((int)(ceil(N / (float)BlockSize)));
	scatter<<<gridSize, BlockSize>>>(Ad, scatterVd, N);
	hipDeviceSynchronize();

	int *scatterV = new int[N];
	int *scatterSum = new int[N+1];
	hipMemcpy(scatterV, scatterVd, N*sizeof(int), hipMemcpyDeviceToHost);
	prefixSumSharedMemWarpper(scatterV, scatterSum, N+1);

	int returnSize = scatterSum[N];

	int *Rd, *Sumd;
	hipMalloc((void**)&Rd, returnSize * sizeof(int));
	hipMalloc((void**)&Sumd, (N+1) * sizeof(int));
	hipMemcpy(Sumd, scatterSum, (N+1)*sizeof(int), hipMemcpyHostToDevice);
	streamCompact<<<gridSize, BlockSize>>>(Ad, Sumd, Rd, N);

	hipEventCreate(&endTime);
	hipEventRecord(endTime, 0);
	hipEventSynchronize(endTime);

	hipEventElapsedTime(&time, startTime, endTime);
	hipEventDestroy(startTime);
	hipEventDestroy(endTime);
	cout << "GPU(Scatter): " << time << " ms" << endl;

	hipMemcpy(R, Rd, returnSize*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(Ad);
	hipFree(scatterVd);
	//hipFree(scatterSum);
	//hipFree(Rd);
}

struct is_not_zero
{
	__host__ __device__
	bool operator()(const int x)
	{
		return (x != 0);
	}
};

void thrustStreamCompaction(int* A, int* &R, int n)
{
	float time;
	hipEvent_t startTime, endTime;
	hipEventCreate(&startTime);
	hipEventRecord(startTime, 0);

	int size = thrust::count_if(A, A+n, is_not_zero());
	R = new int[size];
	thrust::copy_if(A, A+n, R, is_not_zero());

	hipEventCreate(&endTime);
	hipEventRecord(endTime, 0);
	hipEventSynchronize(endTime);

	hipEventElapsedTime(&time, startTime, endTime);
	hipEventDestroy(startTime);
	hipEventDestroy(endTime);
	cout << "GPU(thrust): " << time << " ms" << endl;
}

